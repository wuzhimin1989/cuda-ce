#include "hip/hip_runtime.h"
﻿//#include <stdlib.h>
//#include <stdio.h>
//#include <ctype.h>
//#include <string.h>
#include <iostream>
#include <queue>
#include <iomanip>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <sm_11_atomic_functions.h>
#include ""
using namespace std;

__constant__ int SCCSIZE = 0;
__constant__ int TOTALSIZE = 0;
__constant__ int TASKTYPE = 0;
__constant__ int WARP_T = 32;
__constant__ int BLOCK_T = 512;
__constant__ int INITIAL_T = 32;
__constant__ int EXPAND_LEVEL = 4;
__constant__ int BLOCK_SYN_THRESHOLD = 8;


/***********Template Queue*****************/
template <class Type>  
class QueueItem
{  
public:  
	QueueItem(const Type &t) : item(t), next(0) {}  
	Type item;  
	QueueItem *next;  
};  

template <class Type>  
class Queue
{  
public:  
	Queue() : head(0),tail(0) {this->size = 0}  
	Queue(const Queue &Q):head(0), tail(0)
	{  
		copy_elems(Q);  
		this->size = Q.count();
	}  
	template <class Type2> 
	Queue<Type>& operator=(const Queue<Type2>&);  
	~Queue()
	{ 
		destroy();
	}  
	Type& front() 
	{ 
		return head->item;
	}  
	const Type &front() const { return head->item; }  
	void push( const Type& );  
	void pop();  
	int count();
	bool empty() const
	{  
		return head == 0;  
	}  
private:
	int size;
	QueueItem<Type> *head;  
	QueueItem<Type> *tail;  
	void destroy();  
	void copy_elems(const Queue&);  
}; 

template <class Type>
int Queue<Type>::count()
{
	return this->size;
}

template <class Type>  
void Queue<Type>::push( const Type& val)  
{  
	QueueItem<Type> *pt = new QueueItem<Type>(val);  
	if(empty())  
		head = tail = pt;  
	else {  
		tail->next = pt;  
		tail = pt;  
	} 
	this->size++;
}  

template <class Type>  
void Queue<Type>::pop()  
{  
	QueueItem<Type>* p = head;  
	head = head->next;
	this->size--;
	delete p;  
}  

template <class Type>  
void Queue<Type>::destroy()  
{  
	while(!empty())  
		pop();  
}  

template <class Type>  
void Queue<Type>::copy_elems(const Queue& orig)  
{  
	for(QueueItem<Type>* pt = orig.head;pt;pt = pt->next)  
		push(pt->item);  
} 

template <class Type>  
template <class Type2> 
Queue<Type>& Queue<Type>::operator=( const Queue<Type2>& orig)  
{  
	if((void*)this == (void*)&orig) {  
		*this;  
	}  
	Queue<Type2> tmp(orig);  
	destroy(); // delete  

	while(!tmp.empty())  
	{  
		push(tmp.front());  
		tmp.pop();  
	}  
	this->size = orig.count();
	return *this;  
}  
/********************************************************************/

//class pathnode
class Pathnode{
public:
	int Nid;
	queue<int> ** tmppath;
	int queueindex;

	Pathnode(){}
	Pathnode(int nid)
	{
		Nid = nid;
		if(TASKTYPE == 1)
		{
			tmppath = new queue<int> * [TOTALSIZE-SCCSIZE];
		}
		else
		{
			tmppath = new queue<int>* [SCCSIZE - 1];
		}
		queueindex = 0;
	}

	Pathnode& operator =(const Pathnode &)
	{

	}
};

//class Gqueue for global memeory access
class GQueue{
public:
	Pathnode ** G_queue;
	//Pathnode ** G_Backup_queue;
	int * G_queue_size;
	//int * G_backup_queue_size; //as a backup
	int blockcount;
	//int backupblockcount;

	GQueue()
	{
		blockcount = 0;
		backupblockcount = 0;
	}
	~GQueue(){;}
};


/***************Global variant****************/
__device__ GQueue G_Queue;
__device__ bool G_ifsccReach;
__device__ Pathnode ** P_G_sequence_index; //as a sequencial array to do task partition
__device__ int * P_taskd_index;

//for child use
__device__ Pathnode ** C_G_sequence_index; //as a sequencial array to do task partition
__device__ int * C_taskd_index;

__device__ int Child_Expandedtask;
__device__ bool Child_syn_need;
__device__ bool Child_need_back2parent;

__device__ int * Child_Queue_index;


//for the syn between blocks 
__device__ int SynMutex; //for simple syn  
__device__ int * Arrayin;
__device__ int * Arrayout;


//syn between blocks
__device__ void __gpu_blocks_simple_syn(int goalval)
{
	//thread ID in a block
	int tid_in_block = threadIdx.x * blockDim.y + threadIdx.y;
	
	// only thread 0 is used for synchronization
	if (tid_in_block == 0) 
	{
		atomicAdd(&SynMutex, 1);
			
		//only when all blocks add 1 to g_mutex will
		//g_mutex equal to goalVal
		while(SynMutex != goalval) {
			;
		}
	}
	hipDeviceSynchronize();
}
__device__ void __gpu_blocks_tree_syn(int goalval, int * arrayin, int * arrayout)
{
	// thread ID in a block
	int tid_in_block = threadIdx.x * blockDim.y + threadIdx.y;
	int nBlockNum = gridDim.x * gridDim.y;
	int bid = blockIdx.x * gridDim.y + blockIdx.y;
	
	// only thread 0 is used for synchronization
	if (tid_in_block == 0) 
	{
		Arrayin[bid] = goalval;
	}
	
	if (bid == 1)
	{
		if (tid_in_block < nBlockNum) 
		{
			while (Arrayin[tid_in_block] != goalval) 
			{;}
		}
		hipDeviceSynchronize();
	
		if (tid_in_block < nBlockNum)
		{
			Arrayout[tid_in_block] = goalval;
		}
	}
	
	if (tid_in_block == 0)
	{
		while (Arrayout[bid] != goalval)
		{ ;}
	}
	hipDeviceSynchronize();
}

/*******************************************/

//Quick search for if scc reach
__global__ void BSearchIfreach(bool * theresult, int * searchlist, int size, int key)
{
	int Bslow = 0;
	int Bshigh = size-1;
	int Bsmid = Bslow + (Bshigh-Bslow)/2;

	while(Bslow > Bshigh)
	{
		if(key == searchlist[Bsmid])
		{
			*theresult = true;
		}
		else if(key > searchlist[Bsmid])
		{
			Bslow = Bsmid;
		}
		else
		{
			Bshigh = Bsmid;
		}
		Bsmid = Bslow + (Bshigh-Bslow)/2;
	}
}

__global__ void GPath(int startid, int * scc, int ** outgoing, int * path2scc)
{
	int threadindex = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ Queue<Pathnode> Init_S_queue[8];
	extern __shared__ int queuesize;
	extern __shared__ bool ifexpand;
	extern __shared__ bool ifSccReach;
	extern __shared__ unsigned int path2sccmutex;
	extern __shared__ bool iffinish;

	Pathnode * tmpnode;

	if(threadindex == 0)
	{
		tmpnode = new Pathnode(startid);
		tmpnode->tmppath[tmpnode->queueindex] = new queue<int>();
		tmpnode->tmppath[tmpnode->queueindex]->push(startid);

		tmpnode->queueindex ++;
		Init_S_queue[threadindex % 7].push(*tmpnode);
		queuesize = 1;
		ifexpand = false;
		ifSccReach = false;
		path2sccmutex = 0;
	}

	hipDeviceSynchronize();

	do{
		if(threadindex < queuesize)
		{
			Pathnode * peeknode = &Init_S_queue[threadindex % 7].front();
			
			if(peeknode)
			{
				int succ_num = 0;
				
				//judge if belong to scc(sorted)
				BSearchIfreach(&ifSccReach,scc,SCCSIZE,peeknode->Nid);

				if(ifSccReach == true)
				{
					while(!iffinish)  
					{  
						if(atomicExch(&path2sccmutex, 1))   //use lock to modify the path2scc
						{
							for(int i=0; i< peeknode->tmppath[0]->size();i++)
							{
								path2scc[i] = (peeknode->tmppath[0]->front());
							}
							iffinish = true;
							atomicExch(&path2sccmutex, 0);
						}
					}
					break;
				}

				while(outgoing[peeknode->Nid][succ_num] != -1)
				{					
					int pathcount = 0;
					bool ifnewjudge = true;

					tmpnode = new Pathnode(outgoing[peeknode->Nid][succ_num]);
					while(pathcount < peeknode->queueindex)
					{
						tmpnode->tmppath[tmpnode->queueindex] = peeknode->tmppath[tmpnode->queueindex];
						tmpnode->tmppath[tmpnode->queueindex]->push(tmpnode->Nid);
						tmpnode->queueindex++;
						pathcount ++;
					}
					
					Init_S_queue[threadindex % 7].push(*tmpnode);
					succ_num++;
				}
				Init_S_queue[threadindex % 7].pop();
			}			
		}
		if(threadindex == 0)
			iffinish = false;

		hipDeviceSynchronize();

		if(threadindex== 0)
		{
			for(int i = 0; i < 8; i++)
				queuesize += Init_S_queue[i].count();
			if(queuesize > INITIAL_T)
				ifexpand = true;
		}
		hipDeviceSynchronize();
	}while(ifexpand);

	int expandedtasksize = 0;
	int childbsize = 0;

	if(!ifSccReach && threadindex == 0)
	{
		/*!!!important!!!FOR THIS PART, how many task to put in each block is very important, in order to decrease the time to call child, 
		* maybe the thread in each block should be more than the task, this can be verified in experiments*/

		for(int i=0; i<8; i++)                   
		{
			expandedtasksize += Init_S_queue[i].count();
		}
		childbsize = expandedtasksize / WARP_T + 1;
		G_Queue.G_queue = new Pathnode * [childbsize];
		G_Queue.G_queue_size = new int [childbsize];
		G_Queue.blockcount = childbsize;

		for(int j=0; j<childbsize; j++)
		{
			G_Queue.G_queue[j] = new __device__ Pathnode[TOTALSIZE-SCCSIZE - WARP_T];
			G_Queue.G_queue_size[j] = 0;
		}
		
		int tmpcount = 0;
		for(int j = 0; j < 8; j++)
		{
			for(int m = 0; m < Init_S_queue[j].count() - INITIAL_T/8; m++)
			{
				G_Queue.G_queue[tmpcount][G_Queue.G_queue_size[tmpcount]] = Init_S_queue[j].front();    //not sure about if the memory copy will work,need confirm.
				G_Queue.G_queue_size[tmpcount]++;
				Init_S_queue[j].pop();
				tmpcount++;
				tmpcount=tmpcount%(childbsize-1);
			}
		}
	}

	hipDeviceSynchronize();
	
	int expandtime = 1;
	queuesize = 0;
	while(!G_ifsccReach && !ifSccReach)
	{
		//this can be expanded to two version, one is iterative, the other is recursive!
		bool ifneedsyn = false;    

		//rearrange tasks
		if(threadindex == 0)
		{
			if(expandtime > 1)  
			{
				P_taskd_index = new int[childbsize + 1];   //add 1 is for the end of the last block
				for(int i = 0; i < G_Queue.blockcount + 1; i++)
				{
					queuesize += G_Queue.G_queue_size[i];
					P_taskd_index[i] = queuesize;
				}
			}
			P_G_sequence_index = new __device__ Pathnode * [queuesize];
			expandedtasksize = queuesize;
		}

		if(threadindex < childbsize)
		{
			int beginindex = P_taskd_index[threadindex];
			/*for(int i=0; i<G_Queue.G_queue_size[threadindex]; i++)
			{
			P_G_sequence_index[i+beginindex] = &G_Queue.G_queue[threadindex][i];
			}*/
			//if array are sequential stored then:
			P_G_sequence_index[beginindex] = G_Queue.G_queue[threadindex];
		}
		if(childbsize > BLOCK_T)
		{
			int beginindex, leftsize;
			for(int j = 0; j<childbsize/BLOCK_T; j++)
			{
				beginindex = P_taskd_index[j*BLOCK_T + threadindex];
				P_G_sequence_index[beginindex] = G_Queue.G_queue[j*BLOCK_T + threadindex];
			}
			leftsize = childbsize % BLOCK_T;
			if(threadindex < leftsize)
				P_G_sequence_index[P_taskd_index[(childbsize/BLOCK_T)*BLOCK_T + threadindex]] = G_Queue.G_queue[(childbsize/BLOCK_T)*BLOCK_T + threadindex];
		}
		////////////////////////////////
	
		if(threadindex == 0)
		{
			int averagetask = expandedtasksize/childbsize + 1;
			if(averagetask > WARP_T)
			{
				childbsize = expandedtasksize/WARP_T + 1;
				averagetask = expandedtasksize/childbsize + 1;
			}

			for(int i=0; i<childbsize + 1; i++) 
			{
				P_taskd_index[i] = i*averagetask;
			}

			if(childbsize > 1)
			{
				Arrayin = new __device__ int[childbsize];
				Arrayout = new __device__ int[childbsize];
				ChildPath<<<(EXPAND_LEVEL*(averagetask)), childbsize>>>(P_G_sequence_index, P_taskd_index, path2scc,scc,outgoing);
				hipDeviceSynchronize();
			}
			else
			{
				Arrayin = new __device__ int[1];
				Arrayout = new __device__ int[1];
				ChildPath<<<512,1>>>(P_G_sequence_index, P_taskd_index, path2scc,scc,outgoing);
				hipDeviceSynchronize();
			}
			//call child path,how to combine each block to just one SM?
			
			expandtime++;
			expandedtasksize = 0;
			ifneedsyn = true;
		}

		if(ifneedsyn)
			hipDeviceSynchronize();
	}	
}

__global__ void ChildPath(Pathnode ** G_sequence_Queue, int * taskindex, int * p2scc, int * scc, int ** outgoing)   //dynamic parallel in cuda
{
	int inblocktindex = threadIdx.x;
	int globalthreadindex = blockDim.x * blockIdx.x + threadIdx.x;

	extern __shared__ Queue<Pathnode> Child_Init_S_queue[8];
	extern __shared__ int queuesize;
	extern __shared__ bool ifSccReach;
	extern __shared__ bool iffinish;
	extern __shared__ unsigned int C_path2sccmutex;


	int duration=taskindex[blockIdx.x + 1] - taskindex[blockIdx.x];
	int goalVal = 0;

	Pathnode * Childpeeknode;

	if(inblocktindex == 0)
	{
		queuesize = duration;
		ifSccReach = false;
	}
	if(globalthreadindex == 0)
	{
		Child_syn_need = false;
		Child_need_back2parent = false;
		Child_Queue_index = new __device__ int[gridDim.x];
		Child_Expandedtask = 0;
	}

	if(gridDim.x < BLOCK_SYN_THRESHOLD)
		__gpu_blocks_simple_syn(gridDim.x);
	else
		__gpu_blocks_tree_syn(goalVal++, Arrayin, Arrayout);
	
	while(!G_ifsccReach && !Child_need_back2parent)
	{
		//copy data from global memory to shared memory
		duration=taskindex[blockIdx.x + 1] - taskindex[blockIdx.x];

		if(inblocktindex < duration)
		{
			for(int i=0; i<duration/8; i++)
			{
				Child_Init_S_queue[inblocktindex % 7].push(*(G_sequence_Queue)[taskindex[blockIdx.x]+ i*duration + inblocktindex]);
			}
		}
		if(inblocktindex < duration - (duration/8)*8)
		{
			Child_Init_S_queue[inblocktindex % 7].push(*(G_sequence_Queue)[taskindex[blockIdx.x] + (duration/8)*8 + inblocktindex]);
		}
		hipDeviceSynchronize();

		if(globalthreadindex == 0)   //!not confirmed if needed this
			free(*G_sequence_Queue);
		//////////////////////////////////////////////////
		if(inblocktindex < queuesize)
		{
			Childpeeknode = &Child_Init_S_queue[inblocktindex % 7].front();
			if(Childpeeknode)
			{
				int succ_num = 0;
				BSearchIfreach(&ifSccReach, scc, SCCSIZE, Childpeeknode->Nid);

				if(ifSccReach == true)
				{
					while(!iffinish)  
					{  
						if(atomicExch(&C_path2sccmutex, 1))   //use lock to modify the path2scc
						{
							for(int i=0; i< Childpeeknode->tmppath[0]->size();i++)
							{
								p2scc[i] = (Childpeeknode->tmppath[0]->front());
							}
							iffinish = true;
							atomicExch(&C_path2sccmutex, 0);
						}
					}

					if(inblocktindex == 0)
						G_ifsccReach = true;
					break;
				}

				Pathnode * tmpnode;

				while(outgoing[Childpeeknode->Nid][succ_num] != -1)
				{
					int pathcount = 0;
					bool ifnewjudge = true;

					tmpnode = new Pathnode(outgoing[Childpeeknode->Nid][succ_num]);
					while(pathcount < Childpeeknode->queueindex)
					{
						tmpnode->tmppath[tmpnode->queueindex] = Childpeeknode->tmppath[tmpnode->queueindex];
						tmpnode->tmppath[tmpnode->queueindex]->push(tmpnode->Nid);
						tmpnode->queueindex++;
						pathcount ++;
					}

					Child_Init_S_queue[inblocktindex % 7].push(*tmpnode);
					succ_num++;
				}

				Child_Init_S_queue[inblocktindex % 7].pop();				
			}
		}

		if(inblocktindex == 0)
			iffinish = false;
		hipDeviceSynchronize();

		//calculate queuesize;
		int cpbackindex[8];
		if(inblocktindex == 0)
		{
			for(int i = 0; i < 8; i++)
			{
				queuesize += Child_Init_S_queue[i].count();
				cpbackindex[i]=queuesize;
			}
			if(queuesize > blockDim.x)
				Child_syn_need = true;
			Child_Queue_index[blockIdx.x] = queuesize;
		}

		if(gridDim.x < BLOCK_SYN_THRESHOLD)
			__gpu_blocks_simple_syn(gridDim.x);
		else
			__gpu_blocks_tree_syn(goalVal++, Arrayin, Arrayout);

		if(Child_syn_need)
		{
			if(globalthreadindex == 0)
			{
				int averagetask, lefttask;
				for(int i=0; i<G_Queue.blockcount;i++)
				{
					Child_Expandedtask += G_Queue.G_queue_size[i]; 
				}
				
				if(Child_Expandedtask > (gridDim.x * blockDim.x))
					Child_need_back2parent = true;
				else
				{
					G_sequence_Queue = new Pathnode *[Child_Expandedtask];
					averagetask = Child_Expandedtask/(gridDim.x);
					lefttask = Child_Expandedtask - averagetask*(gridDim.x);
					for(int i=0;i<gridDim.x;i++)
					{
						if(i<lefttask)
							taskindex[i] = averagetask+1;
						else
							taskindex[i] = averagetask;
					}
				}
			}

			if(gridDim.x < BLOCK_SYN_THRESHOLD)
				__gpu_blocks_simple_syn(gridDim.x);
			else
				__gpu_blocks_tree_syn(goalVal++, Arrayin, Arrayout);

			if(blockIdx.x < G_Queue.blockcount)
			{
				if(inblocktindex<8)
				{
					for(int m = 0; m < Child_Init_S_queue[inblocktindex].count(); m++)
					{
						G_Queue.G_queue[blockIdx.x][cpbackindex[inblocktindex]+m] = Child_Init_S_queue[inblocktindex].front();    //not sure about if the memory copy will work,need confirm.
						Child_Init_S_queue[inblocktindex].pop();
					}
				}
				if(inblocktindex == 0)
				{
					G_sequence_Queue[blockIdx.x * queuesize] = G_Queue.G_queue[blockIdx.x];
					G_Queue.G_queue_size[blockIdx.x] = queuesize;
				}
			}
			else
			{
				if(inblocktindex<8)
				{
					for(int m = 0; m< Child_Init_S_queue[inblocktindex].count(); m++)
					{
						int tmp = blockIdx.x % G_Queue.blockcount;
						G_Queue.G_queue[m][Child_Queue_index[m]+cpbackindex[inblocktindex]+m] = Child_Init_S_queue[inblocktindex].front();
						Child_Init_S_queue[inblocktindex].pop();
					}
				}
				if(inblocktindex == 0)
				{
					G_sequence_Queue[blockIdx.x * queuesize] = G_Queue.G_queue[blockIdx.x];
					G_Queue.G_queue_size[blockIdx.x % G_Queue.blockcount] += queuesize;
				}
			}

			if(gridDim.x < BLOCK_SYN_THRESHOLD)
				__gpu_blocks_simple_syn(gridDim.x);
			else
				__gpu_blocks_simple_syn(goalVal++, Arrayin, Arrayout);

		}

	}
}


//Cuda Quicksort
__global__ void Gquicksort(int * data, int left, int right)
{
	int nleft, nright;
	hipStream_t s1, s2;

	partition(data, left, right, &nleft, &nright);

	if(left < nright)
	{
		hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
		Gquicksort<<<1,1,s1>>>(data, left, nright);
	}
	if(nleft < right)
	{
		hipStreamCreateWithFlags(&s2, hipStreamNonBlocking);
		Gquicksort<<<1,1,s2>>>(data, nleft, right);
	}

}
void partition(int* array, int left, int right, int * nleft, int *nright)
{
	int index = left;
	int pivot = array[index];	
	swap(array[index], array[right]);
	for (int i=left; i<right; i++)
	{
		if (array[i] > pivot)    
			swap(array[index++], array[i]);
	}
	swap(array[right], array[index]);
	*nleft = index-1;
	*nright = index+1;
}
////////////////////////

string CudaIsfair(int fairnesstype, int sccsize, int * sccnodelist, int * evetlist, int ** outgoingtrainsition)
{
	TASKTYPE = 0;
}

__global__ void CudaUnion()
{

}



//outtoing array: should add a -1 in the end of each array.
string CudaGetCounterexample(int initial_t, int sccsize, int accsize, int totalsize, int startID, int * sccnodelist, int * acceptlist, int ** outgoingtransition, int outgoingwidth) //sccnodelist and acceptlist should be sorted for quick search
{
	int deviceCount;
	int * G_path2scc, *H_path2scc;
	int * G_path2acc, *H_path2acc;
	int ** G_outgoing;
	int * G_sccnodelist;
	int * G_acceptlist;
	int i;
	size_t acturalsize;

	SCCSIZE = sccsize;
	TOTALSIZE = totalsize;
	TASKTYPE = 1;
	INITIAL_T = initial_t;

	hipGetDeviceCount(&deviceCount);
	if(deviceCount == 0)
		return NULL;

	//cudasetdevice();  //optional to use
	H_path2scc = new int[totalsize-sccsize];
	for(i = 0; i < totalsize-sccsize; i++)
		H_path2scc = -1;
	H_path2scc = new int[sccsize];
	for(i = 0; i < totalsize-sccsize; i++)
		H_path2scc = -1;

	hipMalloc((void**)&G_path2scc, sizeof(int)*(totalsize-sccsize));
	hipMalloc((void**)&G_path2acc, sizeof(int)*sccsize);
	hipMalloc((void**)&G_sccnodelist, sizeof(int)*sccsize);
	hipMalloc((void**)&G_acceptlist, sizeof(int)*accsize); 
	hipMallocPitch((void**)&G_outgoing, &acturalsize, sizeof(int)*outgoingwidth, totalsize);    //outgoing from pat should be a n*m

	hipMemcpy(G_path2scc,H_path2scc,sizeof(int)*(totalsize-sccsize),hipMemcpyHostToDevice);
	hipMemcpy(G_path2acc,H_path2acc,sizeof(int)*(sccsize),hipMemcpyHostToDevice);
	hipMemcpy(G_sccnodelist,sccnodelist,sizeof(int)*sccsize, hipMemcpyHostToDevice);
	hipMemcpy(G_acceptlist,acceptlist,sizeof(int)*sccsize, hipMemcpyHostToDevice);
	hipMemcpy2D(G_outgoing,acturalsize,outgoingtransition,sizeof(int)*outgoingwidth,outgoingwidth,totalsize,hipMemcpyHostToDevice);
	/*hipStream_t counterexampleStream[2];
	hipStreamCreate(&counterexampleStream[0]);
	hipStreamCreate(&counterexampleStream[1]);
	*/
	dim3 blockparameterp(INITIAL_T,1,1);
	dim3 gridparameterp(1,1,1);
	//int gridparameter = 1; optional
	Gquicksort<<<1,1>>>(&sccnodelist, 0, SCCSIZE - 1);
	hipDeviceSynchronize();
	//hipSetDevice();  optional to use
	GPath<<<blockparameterp, gridparameterp, 32>>>(startID, G_sccnodelist, G_outgoing, G_path2scc);
	hipMemcpy(H_path2scc,G_path2scc, sizeof(int)*(totalsize-sccsize), hipMemcpyDeviceToHost);

	GPath<<<blockparameterp,gridparameterp, 32>>>()

}

